#include<stdio.h>
#include<hip/hip_runtime.h>
__global__ void square(int *arr)
{
__shared__ int a[2];
int id=threadIdx.x;
if(id==0)
{
a[0]=arr[0]+arr[2];
}
else
a[1]=arr[2]*arr[2];
__syncthreads();
for(int i=0;i<2;i++)
{
arr[i+3]=a[i];
}
arr[5]=(a[0]*arr[1])+a[1];
}
int main()
{
int n=6,a[n],b[n],no,x,y;
printf("Enter the number:-");
scanf("%d",&no);
if(no<=55)
{
	x=10;
}
else
	x=100;
y=no-x;
a[0]=no;
a[1]=x;
a[2]=y;
printf("\nValue of deviation is: %d",y);
int *d_d;
hipMalloc((void**)&d_d,n*sizeof(int));
hipMemcpy(d_d,a,n*sizeof(int),hipMemcpyHostToDevice);
square <<<1,2>>> (d_d);
hipMemcpy(b,d_d,n*sizeof(int),hipMemcpyDeviceToHost);
printf("\nd_d:%d",d_d);
printf("\nOUTPUT FOR 1ST THREAD(number+deviation):%d\t",b[3]);
printf("\nOUTPUT FOR 2nd THREAD(deviation*deviation):%d\t",b[4]);
printf("\nSQUARE OF %d : %d",no,b[5]);
hipFree(d_d);
return 0;
}
